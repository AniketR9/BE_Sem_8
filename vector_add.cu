
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
#include<chrono>
using namespace std;
using namespace std::chrono;

__global__ void vecAdd(int *a, int *b, int *c, int N){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<N){
        c[tid]=a[tid]+b[tid];
    }
}

void printVec(vector<int> &vc){
    for(auto x:vc){
        cout<<x<<" ";
    }
    cout<<endl;
}

int main(){
    int size=4;
    vector<int> a={1,2,3,4};
    vector<int> b={5,6,7,8};
    vector<int> c(size);
    vector<int> c1(size);

    auto start1=high_resolution_clock::now();
    for(int i=0;i<size;i++){
        c[i]=a[i]+b[i];
    }
    auto end1=high_resolution_clock::now();
    cout<<"Vec A : ";printVec(a);
    cout<<"Vec B : ";printVec(b);
    cout<<"Vec C : ";printVec(c);
    duration<double,milli> timet=end1-start1;
    cout<<"Time for sequential : "<<timet.count()<<" ms."<<endl;

    int *da, *db,*dc;
    size_t bytes=size*sizeof(int);

    hipMalloc(&da,bytes);
    hipMalloc(&db, bytes);
    hipMalloc(&dc,bytes);

    hipMemcpy(da,a.data(),bytes, hipMemcpyHostToDevice);
    hipMemcpy(db,b.data(),bytes, hipMemcpyHostToDevice);

    int threads=256;
    int blocks=(size+threads-1)/threads;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    vecAdd<<<blocks,threads>>>(da,db,dc,size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_vec_time;
    hipEventElapsedTime(&gpu_vec_time, start,stop);
    hipMemcpy(c1.data(),dc,bytes,hipMemcpyDeviceToHost);
    cout<<"Vector C1 : ";printVec(c1);
    cout<<"Time taken for parallel : "<<gpu_vec_time<<"ms\n";

    hipFree(da);hipFree(db),hipFree(dc);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    const int n1=1<<20; //1 million elements;
    vector<int> a1(n1,1);
    vector<int> a2(n1,2);
    vector<int> c2(n1);
    vector<int> c3(n1);

    auto start_time=chrono::high_resolution_clock::now();
    for(int i=0;i<n1;i++){
        c2[i]=a1[i]+a2[i];
    }
    auto end_time=chrono::high_resolution_clock::now();
    chrono::duration<double,milli> cpu_vec_time=end_time-start_time;
    cout<<"Sequential Execution Time Required: "<<cpu_vec_time.count()<<" ms\n";

    // int* da,*db,*dc;
    bytes=n1*sizeof(int);

    hipMalloc(&da,bytes);
    hipMalloc(&db,bytes);
    hipMalloc(&dc, bytes);

    hipMemcpy(da,a1.data(),bytes,hipMemcpyHostToDevice);
    hipMemcpy(db,a2.data(),bytes,hipMemcpyHostToDevice);

    // int threads=256;
    blocks=(n1+threads-1)/threads;
    hipEvent_t start2,stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    vecAdd<<<blocks,threads>>>(da,db,dc,n1);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);

    float gpu_vec_time1=0;
    hipEventElapsedTime(&gpu_vec_time1,start2,stop2);
    hipMemcpy(c3.data(),dc,bytes,hipMemcpyDeviceToHost);

    cout<<"Parallel Execution Time Required: "<<gpu_vec_time<<" ms\n";
    hipFree(da);hipFree(db);hipFree(dc);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}