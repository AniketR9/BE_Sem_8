#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<chrono>
using namespace std;
using namespace std::chrono;

__global__ void matmul(int *a , int *b, int *c, int N){
    int row=blockIdx.y*blockDim.y+threadIdx.y;
    int col=blockIdx.x*blockDim.x+threadIdx.x;

    if(row<N && col<N){
        int sum=0;
        for(int i=0;i<N;i++){
            sum+=a[row*N+k]*b[k*N+col];
        }
        c[row*N+col]=sum;
    }
}

void printMat(vector<int> vc, int N){
    for(int i=0;i<N*N;i++){
        cout<<vc[i]<<" ";
        if((i+1)%N==0)cout<<endl;
    }
    cout<<endl;
}

int main(){
    const int n1=3;
    vector<int> a={1,2,3,4,5,6,7,8,9};
    vector<int> b={9,8,7,6,5,4,3,2,1};
    vector<int> c(n1*n1);
    vector<int> c_seq(n1*n1);

    auto begint=high_resolution_clock::now();
    for(int i=0;i<n1;i++){
        for(int j=0;j<n1;j++){
            int sum=0;
            for(int k=0;k<n1;k++){
                sum+=a[i*n1+k]*b[k*n1+j];
            }
            c_seq[i*n1+j]=sum;
        }
    }
    auto endt=high_resolution_clock::now();
    duration<double, milli> timet1=endt-begint;
    cout<<"Mat A:\n ";printMat(a,n1);
    cout<<"Mat B:\n ";printMat(b,n1);
    cout<<"Mat C:\n ";printMat(c,n1);
    cout<<"Time for seq exec:"<<timet1.count()<<" ms.\n";

    int *da,*db,*dc;
    size_t bytes=n1*n1*sizeof(int);

    hipMalloc(&da,bytes);
    hipMalloc(&db,bytes);
    hipMalloc(&dc,bytes);

    hipMemcpy(da,a.data(),bytes,hipMemcpyHostToDevice);
    hipMemcpy(db,b.data(),bytes,hipMemcpyHostToDevice);

    dim3 threadsPerBlock(n1,n1);
    dim3 blocksPerGrid(1,1);

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matmul<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,n1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(c.data(),dc, bytes, hipMemcpyDeviceToHost);
    cout<<"Matrix C:\n";printMat(c);
    float gpu_time;
    hipEventElapsedTime(&gpu_time,start,stop);
    hipFree(da);hipFree(db);hipFree(dc);
    hipEventDestroy(start);
hipEventDestroy(stop);

const int N = 512; // 512x512 matrix
    vector<int> matA(N * N, 1);
    vector<int> matB(N * N, 2);
    vector<int> matC(N * N, 0);
    vector<int> matC_seq(N * N, 0);

    // Sequential Matrix Multiplication
    auto start_cpu = chrono::high_resolution_clock::now();
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            int sum = 0;
            for (int k = 0; k < N; ++k)
                sum += matA[row * N + k] * matB[k * N + col];
            matC_seq[row * N + col] = sum;
        }
    }
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> cpu_mat_time = end_cpu - start_cpu;
    cout << "[CPU] Matrix Mul Time: " << cpu_mat_time.count() << " ms\n";

    // Parallel Matrix Multiplication
    int *d_matA, *d_matB, *d_matC;
    size_t matrixBytes = N * N * sizeof(int);
    hipMalloc(&d_matA, matrixBytes);
    hipMalloc(&d_matB, matrixBytes);
    hipMalloc(&d_matC, matrixBytes);

    hipMemcpy(d_matA, matA.data(), matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB.data(), matrixBytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_matA, d_matB, d_matC, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_mat_time = 0;
    hipEventElapsedTime(&gpu_mat_time, start, stop);

    hipMemcpy(matC.data(), d_matC, matrixBytes, hipMemcpyDeviceToHost);
    cout << "[GPU] Matrix Mul Time: " << gpu_mat_time << " ms\n";

    hipFree(d_matA); hipFree(d_matB); hipFree(d_matC);

    hipEventDestroy(start);
    hipEventDestroy(stop);
return 0;
}
